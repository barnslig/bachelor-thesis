#include <cstdio>
#include <random>

#include "CudaHelper.cuh"
#include "Grapple.cuh"

/**
 * Number of Grapple runs
 *
 * Each block is a VT and as we have 250 (kGrappleVTs) blocks per run, we just
 * divide by this.
 */
// TODO does this make sense? We still find way less waypoints than expected
constexpr int kGrappleRuns = 50000 / kGrappleVTs;

int main()
{
  printf("run, block, thread, state\n");

  // std::random_device rd; // Will be used to obtain a seed for the random number engine
  unsigned int seed = 1736331306; // rd(); // Use a constant seed for reproducible results
  std::mt19937 gen(seed);         // Standard mersenne_twister_engine seeded with rd()

  hipStream_t stream[kGrappleRuns];
  int ret = 0;

  for (int i = 0; i < kGrappleRuns; i += 1)
  {
    /* Each Grapple run gets assigned to a different CUDA stream to achieve
     * maximum concurrency
     */
    hipStreamCreate(&stream[i]);

    ret = runGrapple(i, State{0}, &gen, &stream[i]);
    if (ret != 0)
    {
      // Terminate program execution when a single Grapple run has failed
      goto terminate;
    }
  }

terminate:

  // Wait for all CUDA streams to terminate
  gpuErrchk(hipDeviceSynchronize());

  // Check that the kernel launch was successful
  gpuErrchk(hipGetLastError());

  // Final cleanup of the device before we leave
  gpuErrchk(hipDeviceReset());

  return 0;
}
