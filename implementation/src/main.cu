#include <unistd.h>

#include <cstdio>
#include <iostream>
#include <random>

#include "CudaHelper.cuh"
#include "Grapple.cuh"

/**
 * Default number of Grapple runs
 */
constexpr int kDefaultRuns = 50000;

int main(int argc, char *const argv[])
{
  std::random_device rd;

  int argRuns = kDefaultRuns;
  int argSeed = rd(); // 1736331306

  int c;
  while ((c = getopt(argc, argv, "s:n:h")) != -1)
  {
    switch (c)
    {
    case 's':
      argSeed = std::strtol(optarg, nullptr, 10);
      break;
    case 'n':
      /* Each block is a VT and as we have 250 (kGrappleVTs) blocks per
       * run, we just divide by this.
       */
      // TODO does this make sense? We still find way less waypoints than expected
      argRuns = std::strtol(optarg, nullptr, 10) / 250;
      break;
    case '?':
    case 'h':
      std::cerr << "Usage: " << argv[0] << " [options]\n\n";
      std::cerr << "Option        Description\n";
      std::cerr << " -s <seed>    Seed used for hash function diversification. Default: Random number\n";
      std::cerr << " -n <runs>    Number of Grapple runs. Default: " << kDefaultRuns << "\n";
      std::cerr << " -h           Show this message\n";
      exit(EXIT_FAILURE);
    }
  }

  printf("run, block, thread, state\n");

  std::mt19937 gen(argSeed);

  hipStream_t stream[argRuns];
  int ret = 0;

  for (int i = 0; i < argRuns; i += 1)
  {
    /* Each Grapple run gets assigned to a different CUDA stream to achieve
     * maximum concurrency
     */
    hipStreamCreate(&stream[i]);

    ret = runGrapple(i, State{0}, &gen, &stream[i]);
    if (ret != 0)
    {
      // Terminate program execution when a single Grapple run has failed
      goto terminate;
    }
  }

terminate:

  // Wait for all CUDA streams to terminate
  gpuErrchk(hipDeviceSynchronize());

  // Check that the kernel launch was successful
  gpuErrchk(hipGetLastError());

  // Final cleanup of the device before we leave
  gpuErrchk(hipDeviceReset());

  return 0;
}
